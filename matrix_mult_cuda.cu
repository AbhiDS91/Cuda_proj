#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define N 512

// CUDA Kernel
__global__ void matMul(int* A, int* B, int* C, int n) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int size = N * N * sizeof(int);
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;

    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);

    // Init matrices
    for (int i = 0; i < N * N; ++i) {
        A[i] = 1;
        B[i] = 1;
    }

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);

    auto start = std::chrono::high_resolution_clock::now();

    matMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();
    float duration = std::chrono::duration<float, std::milli>(end - start).count();
    std::cout << "GPU Matrix multiplication took " << duration << " ms\n";

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    std::cout << "C[0][0] = " << C[0] << "\n";

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(A); free(B); free(C);

    return 0;
}
